#include "hip/hip_runtime.h"
﻿#include	<stdio.h>
#include	<malloc.h>
#include	<stdlib.h>
#include	<cutil_inline.h>

// 行列のサイズ、ブロックサイズ
#define	MATRIX_SIZE	4096
#define	BLOCK_SIZE	128

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int
main(int argc, char** argv)
{
	// 変数宣言
	unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE;
	int* hMatrixA;
	int* hMatrixB;
	int* hMatrixC;
	hMatrixA = (int*) malloc(matrixSize);
	hMatrixB = (int*) malloc(matrixSize);

	// 初期値設定
	unsigned int	col_idx, row_idx;
	for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
		for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
			hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
			hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
		}
	}

	// デバイス側の変数
	int* dMatrixA;
	int* dMatrixB;
	int* dMatrixC;

	// デバイスメモリの確保
	cutilSafeCall(hipMalloc((void**) &dMatrixA, matrixSize));
	cutilSafeCall(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMalloc((void**) &dMatrixB, matrixSize));
	cutilSafeCall(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMalloc((void**) &dMatrixC, matrixSize));

	// ブロックサイズとグリッドサイズの設定
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);

	// タイマー変数の宣言、測定開始
	printf("Matrix calculation start in the GPU!\n");
	printf("Matrix size\t:\t%d * %d\n", MATRIX_SIZE, MATRIX_SIZE);
	printf("BlockSize\t:\t%d\nGridSize\t:\t%d\n", BLOCK_SIZE, grid);
	float millseconds = 0.0f;
	float sum = 0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < 10; i++){
		hipEventRecord(start, 0);

		// カーネルの起動
		matrixMul <<<grid, block>>>(dMatrixA, dMatrixB, dMatrixC);
		hipDeviceSynchronize();

		// 測定終了、結果表示
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&millseconds, start, stop);
		printf("Time required\t:\t%f millseconds\n", millseconds);
		sum += millseconds;
	}
//	printf("Matrix size\t:\t%d * %d\n", MATRIX_SIZE, MATRIX_SIZE);
//	printf("BlockSize\t:\t%d\nGridSize\t:\t%d\n", BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);
	printf("Time average\t:\t%f millseconds\n", sum /10);

	// 結果の領域の確保と、デバイス側からのメモリ転送
	hMatrixC = (int*) malloc(matrixSize);
	cutilSafeCall(hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost));

	// メモリ開放
	free(hMatrixA);
	free(hMatrixB);
	free(hMatrixC);
	cutilSafeCall(hipFree(dMatrixA));
	cutilSafeCall(hipFree(dMatrixB));
	cutilSafeCall(hipFree(dMatrixC));

	hipDeviceReset();
}

// 行列計算をするカーネル関数
__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC)
{
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int scan_idx;
	int	target = 0;

	for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
		// 対象となる部分をかけたものを足していく
		target += inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
		__syncthreads();	// スレッド同期
	}
	inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target;
}
