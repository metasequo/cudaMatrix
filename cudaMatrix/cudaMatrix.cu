#include "hip/hip_runtime.h"
#include	<stdio.h>
#include	<malloc.h>
#include	<stdlib.h>
#include	<cutil_inline.h>

// �s��̃T�C�Y�A�u���b�N�T�C�Y
#define	MATRIX_SIZE	4096
#define	BLOCK_SIZE	128

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int
main(int argc, char** argv)
{
	// �ϐ��錾
	unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE;
	int* hMatrixA;
	int* hMatrixB;
	int* hMatrixC;
	hMatrixA = (int*) malloc(matrixSize);
	hMatrixB = (int*) malloc(matrixSize);

	// �����l�ݒ�
	unsigned int	col_idx, row_idx;
	for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
		for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
			hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
			hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
		}
	}

	// �f�o�C�X���̕ϐ�
	int* dMatrixA;
	int* dMatrixB;
	int* dMatrixC;

	// �f�o�C�X�������̊m��
	cutilSafeCall(hipMalloc((void**) &dMatrixA, matrixSize));
	cutilSafeCall(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMalloc((void**) &dMatrixB, matrixSize));
	cutilSafeCall(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMalloc((void**) &dMatrixC, matrixSize));

	// �u���b�N�T�C�Y�ƃO���b�h�T�C�Y�̐ݒ�
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);

	// �^�C�}�[�ϐ��̐錾�A����J�n
	printf("Matrix calculation start in the GPU!\n");
	printf("Matrix size\t:\t%d * %d\n", MATRIX_SIZE, MATRIX_SIZE);
	printf("BlockSize\t:\t%d\nGridSize\t:\t%d\n", BLOCK_SIZE, grid);
	float millseconds = 0.0f;
	float sum = 0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < 10; i++){
		hipEventRecord(start, 0);

		// �J�[�l���̋N��
		matrixMul <<<grid, block>>>(dMatrixA, dMatrixB, dMatrixC);
		hipDeviceSynchronize();

		// ����I���A���ʕ\��
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&millseconds, start, stop);
		printf("Time required\t:\t%f millseconds\n", millseconds);
		sum += millseconds;
	}
//	printf("Matrix size\t:\t%d * %d\n", MATRIX_SIZE, MATRIX_SIZE);
//	printf("BlockSize\t:\t%d\nGridSize\t:\t%d\n", BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);
	printf("Time average\t:\t%f millseconds\n", sum /10);

	// ���ʂ̗̈�̊m�ۂƁA�f�o�C�X������̃������]��
	hMatrixC = (int*) malloc(matrixSize);
	cutilSafeCall(hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost));

	// �������J��
	free(hMatrixA);
	free(hMatrixB);
	free(hMatrixC);
	cutilSafeCall(hipFree(dMatrixA));
	cutilSafeCall(hipFree(dMatrixB));
	cutilSafeCall(hipFree(dMatrixC));

	hipDeviceReset();
}

// �s��v�Z������J�[�l���֐�
__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC)
{
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int scan_idx;
	int	target = 0;

	for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
		// �ΏۂƂȂ镔�������������̂𑫂��Ă���
		target += inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
		__syncthreads();	// �X���b�h����
	}
	inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target;
}
